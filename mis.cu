#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 1024

__global__ void initArray(int* Array, int value) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	Array[tid] = value;
}

void initializeDevice(int*& G, int*& G_dev, int*& I, int*& I_dev, int*&H, int n) {
	hipMalloc((void**)&I_dev, n * sizeof(int));
	hipMalloc((void**)&G_dev, n * n * sizeof(int));
	hipMalloc((void**)&H, n * sizeof(int));

	initArray<<<n, BLOCK_SIZE>>>(H, 1);

	hipMemcpy(G_dev, G, n * n * sizeof(int), hipMemcpyHostToDevice);

	hipMemset(I_dev, 0, n * sizeof(int));
}

bool findI(int *G, int *I, int n) {
	// G: Adj
	// I: MIS
	// n: N
	// G_dev: AdjDev
	// I_dev: MISDev
	// H: currentGraph
	// K: WithHeavySubset
	// M: HeavySet
	// T: ScoreSet
	// S: IndSet
	// D: Degrees

	int *G_dev, *I_dev, *H;
    initializeDevice(G, G_dev, I, I_dev, H, n);
	int *K, *M, *T, *S, *D;

	unsigned int* RandomChoice;
	hiprandGenerator_t rand_gen;
	hiprandCreateGenerator(&rand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(rand_gen, 1234ULL);

	int currentGSize = n;
	while(currentGSize > 0) {
		hipMemset(M, 0, n * sizeof(int));
		hipMemset(T, 0, n * sizeof(int));
		hipMemset(D, 0, n * sizeof(int));

		int currentHSize = findM(G, H, D, M, currentGSize);
		
		scoreFind(G, K, M, currentHSize, T, RandomChoice, rand_gen);		//O(log N)
		
		hiprandGenerate(rand_gen, RandomChoice, n);
		S = T;
		// indFind << < ceilNN, blockSize >> > (N, IndSet, Adj, RandomChoice);		//O(1)
		// updateWithInd << <ceilN, blockSize >> > (N, MISDev, CurrentGraph, IndSet);	//O(1)
		// updateWithNeighs << <ceilNN, blockSize >> > (N, CurrentGraph, Adj, IndSet);	

		// int newCardinality = countLeft(CurrentGraph);
		// if (newCardinality == currentCardinality) break;
		// currentCardinality = newCardinality;

	}

	int* BrutalChosen = T;	//steal memory
	while (currentGSize) {
	// 	checkError(hipMemcpy(BrutalChosen, CurrentGraph, N * sizeof(int), hipMemcpyDeviceToDevice), __LINE__);
	// 	checkCurandError(hiprandGenerate(rand_gen, RandomChoice, N), __LINE__);
	// 	indFind << < ceilNN, blockSize >> > (N, BrutalChosen, Adj, RandomChoice);	//O(1)
	// 	updateWithInd << <ceilN, blockSize >> > (N, MISDev, CurrentGraph, BrutalChosen);	//O(1)
	// 	updateWithNeighs << <ceilNN, blockSize >> > (N, CurrentGraph, Adj, BrutalChosen);	//O(1)
	// 	hipDeviceSynchronize();
	// 	currentCardinality = countLeft(CurrentGraph);
	}

	hipMemcpy(I, I_dev, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(G_dev);
	hipFree(I_dev);
	hipFree(H);
	hipFree(K);
	hipFree(M);
	hipFree(T);
	hipFree(S);
	hipFree(D);
	hipFree(RandomChoice);

	return true;
}

int findM(int *&G, int *&H, int *&D, int *&M, int currentGSize) {

}

void scoreFind(int*& Adj, int*& WithHeavySubset, int*& HeavySet, int heavySetCardinality, int*& ScoreSet, unsigned int*& RandomChoice, hiprandGenerator_t& rand_gen) {}